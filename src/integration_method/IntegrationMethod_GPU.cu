/***************************************************************************
 *                           IntegratoinMethod_GPU.cu                      *
 *                           -------------------                           *
 * copyright            : (C) 2013 by Francisco Naveros                    *
 * email                : fnaveros@atc.ugr.es                              *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

#include "../../include/integration_method/IntegrationMethod_GPU.h"
#include "../../include/integration_method/IntegrationMethod_GPU2.h"
#include "../../include/neuron_model/TimeDrivenNeuronModel_GPU.h"

//Library for CUDA
#include <hip/hip_runtime_api.h>


IntegrationMethod_GPU::IntegrationMethod_GPU(char * integrationMethodType, int N_neuronStateVariables, int N_differentialNeuronState, int N_timeDependentNeuronState):N_NeuronStateVariables(N_neuronStateVariables), N_DifferentialNeuronState(N_differentialNeuronState), N_TimeDependentNeuronState(N_timeDependentNeuronState){
	IntegrationMethodType=new char [strlen(integrationMethodType)];
	strncpy(IntegrationMethodType,integrationMethodType,strlen(integrationMethodType));
}

IntegrationMethod_GPU::~IntegrationMethod_GPU(){
	delete [] IntegrationMethodType;
	hipFree(Buffer_GPU);
}

char * IntegrationMethod_GPU::GetType(){
	return this->IntegrationMethodType;
}

ostream & IntegrationMethod_GPU::PrintInfo(ostream & out){
	out << "Integration Method Type: " << this->GetType() << endl;

	return out;
}	
		


