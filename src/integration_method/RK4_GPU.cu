#include "hip/hip_runtime.h"
/***************************************************************************
 *                           RK4_GPU.cu                                    *
 *                           -------------------                           *
 * copyright            : (C) 2013 by Francisco Naveros                    *
 * email                : fnaveros@atc.ugr.es                              *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

#include "../../include/integration_method/RK4_GPU.h"
#include "../../include/integration_method/RK4_GPU2.h"
#include "../../include/neuron_model/TimeDrivenNeuronModel_GPU2.h"

//Library for CUDA
#include <hip/hip_runtime_api.h>


RK4_GPU::RK4_GPU(int N_neuronStateVariables, int N_differentialNeuronState, int N_timeDependentNeuronState):IntegrationMethod_GPU("RK4", N_neuronStateVariables, N_differentialNeuronState, N_timeDependentNeuronState){
}

RK4_GPU::~RK4_GPU(){
	hipFree(AuxNeuronState);
	hipFree(AuxNeuronState1);
	hipFree(AuxNeuronState2);
	hipFree(AuxNeuronState3);
	hipFree(AuxNeuronState4);
}

__global__ void RK4_GPU_position(void ** vector, float * element0, float * element1, float * element2, float * element3, float * element4){
	vector[0]=element0;
	vector[1]=element1;
	vector[2]=element2;
	vector[3]=element3;
	vector[4]=element4;
}

void RK4_GPU::InitializeMemoryGPU(int N_neurons, int Total_N_thread){
	int size=5*sizeof(float *);
	hipMalloc((void **)&Buffer_GPU, size);

	hipMalloc((void**)&AuxNeuronState, N_NeuronStateVariables*Total_N_thread*sizeof(float));
	hipMalloc((void**)&AuxNeuronState1, N_NeuronStateVariables*Total_N_thread*sizeof(float));
	hipMalloc((void**)&AuxNeuronState2, N_NeuronStateVariables*Total_N_thread*sizeof(float));
	hipMalloc((void**)&AuxNeuronState3, N_NeuronStateVariables*Total_N_thread*sizeof(float));
	hipMalloc((void**)&AuxNeuronState4, N_NeuronStateVariables*Total_N_thread*sizeof(float));

	RK4_GPU_position<<<1,1>>>(Buffer_GPU, AuxNeuronState, AuxNeuronState1, AuxNeuronState2, AuxNeuronState3, AuxNeuronState4);
}
		





