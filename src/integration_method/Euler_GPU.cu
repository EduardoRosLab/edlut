#include "hip/hip_runtime.h"
/***************************************************************************
 *                           Euler_GPU.cu                                  *
 *                           -------------------                           *
 * copyright            : (C) 2013 by Francisco Naveros                    *
 * email                : fnaveros@atc.ugr.es                              *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

#include "../../include/integration_method/Euler_GPU.h"
#include "../../include/integration_method/Euler_GPU2.h"
#include "../../include/neuron_model/TimeDrivenNeuronModel_GPU2.h"

//Library for CUDA
#include <cutil_inline.h>


Euler_GPU::Euler_GPU(int N_neuronStateVariables, int N_differentialNeuronState, int N_timeDependentNeuronState):IntegrationMethod_GPU("Euler", N_neuronStateVariables, N_differentialNeuronState, N_timeDependentNeuronState){
}

Euler_GPU::~Euler_GPU(){
	hipFree(AuxNeuronState);
}

__global__ void Euler_GPU_position(void ** vector, float * element0){
	vector[0]=element0;
}

void Euler_GPU::InitializeMemoryGPU(int N_neurons, int Total_N_thread){
	int size=1*sizeof(float *);

	hipMalloc((void **)&Buffer_GPU, size);

	hipMalloc((void**)&AuxNeuronState, N_NeuronStateVariables*Total_N_thread*sizeof(float));


	Euler_GPU_position<<<1,1>>>(Buffer_GPU, AuxNeuronState);
}
		





