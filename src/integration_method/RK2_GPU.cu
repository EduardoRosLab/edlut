#include "hip/hip_runtime.h"
/***************************************************************************
 *                           RK2_GPU.cu                                    *
 *                           -------------------                           *
 * copyright            : (C) 2013 by Francisco Naveros                    *
 * email                : fnaveros@atc.ugr.es                              *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

#include "../../include/integration_method/RK2_GPU.h"
#include "../../include/integration_method/RK2_GPU2.h"
#include "../../include/neuron_model/TimeDrivenNeuronModel_GPU2.h"

//Library for CUDA
#include <hip/hip_runtime_api.h>




RK2_GPU::RK2_GPU(int N_neuronStateVariables, int N_differentialNeuronState, int N_timeDependentNeuronState):IntegrationMethod_GPU("RK2", N_neuronStateVariables, N_differentialNeuronState, N_timeDependentNeuronState){
}

RK2_GPU::~RK2_GPU(){
	hipFree(AuxNeuronState);
	hipFree(AuxNeuronState1);
	hipFree(AuxNeuronState2);
}

__global__ void RK2_GPU_position(void ** vector, float * element0, float * element1, float * element2){
	vector[0]=element0;
	vector[1]=element1;
	vector[2]=element2;
}
	
void RK2_GPU::InitializeMemoryGPU(int N_neurons, int Total_N_thread){
	int size=3*sizeof(float *);

	hipMalloc((void **)&Buffer_GPU, size);

	hipMalloc((void**)&AuxNeuronState, N_NeuronStateVariables*Total_N_thread*sizeof(float));
	hipMalloc((void**)&AuxNeuronState1, N_NeuronStateVariables*Total_N_thread*sizeof(float));
	hipMalloc((void**)&AuxNeuronState2, N_NeuronStateVariables*Total_N_thread*sizeof(float));

	RK2_GPU_position<<<1,1>>>(Buffer_GPU, AuxNeuronState, AuxNeuronState1, AuxNeuronState2);
}





