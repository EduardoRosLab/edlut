/***************************************************************************
 *                           TimeEventAllNeurons_GPU.cpp                   *
 *                           -------------------                           *
 * copyright            : (C) 2013 by Francisco Naveros                    *
 * email                : fnaveros@atc.ugr.es                              *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/


#include "../../include/simulation/TimeEventAllNeurons_GPU.h"
#include "../../include/simulation/Simulation.h"
#include "../../include/simulation/EventQueue.h"

#include "../../include/neuron_model/NeuronModel.h"
#include "../../include/neuron_model/TimeDrivenNeuronModel_GPU.h"
#include "../../include/neuron_model/VectorNeuronState.h"

#include "../../include/spike/TimeDrivenInternalSpike.h"
#include "../../include/spike/Network.h"
#include "../../include/spike/Neuron.h"

#include "../../include/simulation/SynchronizeSimulationEvent.h"

#include "../../include/openmp/openmp.h"

		

TimeEventAllNeurons_GPU::TimeEventAllNeurons_GPU(double NewTime, TimeDrivenNeuronModel_GPU * newNeuronModel, Neuron ** newNeurons, Simulation * CurrentSimulation) : Event(NewTime), neuronModel(newNeuronModel), neurons(newNeurons){
	for(int i=0; i<CurrentSimulation->GetNumberOfQueues(); i++){
		SynchronizeSimulationEvent * NewEvent = new SynchronizeSimulationEvent(NewTime);
		CurrentSimulation->GetQueue()->InsertEvent(i,NewEvent);
	}
}

TimeEventAllNeurons_GPU::~TimeEventAllNeurons_GPU(){

}


//Optimized version which executes the internal spikes instead of insert them in the queue.
void TimeEventAllNeurons_GPU::ProcessEvent(Simulation * CurrentSimulation , volatile int * RealTimeRestriction){

	double CurrentTime = this->GetTime();

	if(*RealTimeRestriction<3){
		VectorNeuronState * State=neuronModel->GetVectorNeuronState();

		neuronModel->UpdateState(-1, State, CurrentTime);

		TimeDrivenInternalSpike NewEvent(CurrentTime, State, neurons);
		NewEvent.ProcessEvent(CurrentSimulation, RealTimeRestriction);
	}

	//Next TimeEvent for all cell
	CurrentSimulation->GetQueue()->InsertEventWithSynchronization(new TimeEventAllNeurons_GPU(CurrentTime + neuronModel->TimeDrivenStep_GPU, GetModel(), GetNeurons(), CurrentSimulation));
}

//Optimized version which executes the internal spikes instead of insert them in the queue.
void TimeEventAllNeurons_GPU::ProcessEvent(Simulation * CurrentSimulation){

	double CurrentTime = this->GetTime();

	VectorNeuronState * State=neuronModel->GetVectorNeuronState();

	neuronModel->UpdateState(-1, State, CurrentTime);

	TimeDrivenInternalSpike NewEvent(CurrentTime, State, neurons);
	NewEvent.ProcessEvent(CurrentSimulation);

	//Next TimeEvent for all cell
	CurrentSimulation->GetQueue()->InsertEventWithSynchronization(new TimeEventAllNeurons_GPU(CurrentTime + neuronModel->TimeDrivenStep_GPU, GetModel(), GetNeurons(), CurrentSimulation));
}

TimeDrivenNeuronModel_GPU * TimeEventAllNeurons_GPU::GetModel(){
	return neuronModel;
}

Neuron ** TimeEventAllNeurons_GPU::GetNeurons(){
	return neurons;
}

void TimeEventAllNeurons_GPU::PrintType(){
	cout<<"TimeEventAllNeurons_GPU"<<endl;
}

int TimeEventAllNeurons_GPU::ProcessingPriority(){
	return 7;
}