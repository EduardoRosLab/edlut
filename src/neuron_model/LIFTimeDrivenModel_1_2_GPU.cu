#include "hip/hip_runtime.h"
/***************************************************************************
 *                           LIFTimeDrivenModel_1_2_GPU.cu                 *
 *                           -------------------                           *
 * copyright            : (C) 2012 by Francisco Naveros                    *
 * email                : fnaveros@atc.ugr.es                              *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

#include "../../include/neuron_model/LIFTimeDrivenModel_1_2_GPU.h"
#include "../../include/neuron_model/LIFTimeDrivenModel_1_2_GPU2.h"
#include "../../include/neuron_model/VectorNeuronState.h"
#include "../../include/neuron_model/VectorNeuronState_GPU.h"

#include <iostream>
#include <cmath>
#include <string>

#include "../../include/spike/EDLUTFileException.h"
#include "../../include/spike/Neuron.h"
#include "../../include/spike/InternalSpike.h"
#include "../../include/spike/PropagatedSpike.h"
#include "../../include/spike/Interconnection.h"

#include "../../include/simulation/Utils.h"

#include "../../include/openmp/openmp.h"

#include "../../include/hipError_t.h"
//Library for CUDA
#include <hip/hip_runtime_api.h>

void LIFTimeDrivenModel_1_2_GPU::LoadNeuronModel(string ConfigFile) throw (EDLUTFileException){
	FILE *fh;
	long Currentline = 0L;
	fh=fopen(ConfigFile.c_str(),"rt");
	if(fh){
		Currentline=1L;
		skip_comments(fh,Currentline);
		if(fscanf(fh,"%f",&this->eexc)==1){
			skip_comments(fh,Currentline);

			if (fscanf(fh,"%f",&this->einh)==1){
				skip_comments(fh,Currentline);

				if(fscanf(fh,"%f",&this->erest)==1){
					skip_comments(fh,Currentline);

					if(fscanf(fh,"%f",&this->vthr)==1){
						skip_comments(fh,Currentline);

						if(fscanf(fh,"%f",&this->cm)==1){
							skip_comments(fh,Currentline);

							if(fscanf(fh,"%f",&this->texc)==1){
								skip_comments(fh,Currentline);

								if(fscanf(fh,"%f",&this->tinh)==1){
									skip_comments(fh,Currentline);

									if(fscanf(fh,"%f",&this->tref)==1){
										skip_comments(fh,Currentline);

										if(fscanf(fh,"%f",&this->grest)==1){
											skip_comments(fh,Currentline);

											this->InitialState = (VectorNeuronState_GPU *) new VectorNeuronState_GPU(3);
//NEW CODE------------------------------------------------------------------------------
											} else {
											throw EDLUTFileException(13,60,3,1,Currentline);
										}
									} else {
										throw EDLUTFileException(13,61,3,1,Currentline);
									}
								} else {
									throw EDLUTFileException(13,62,3,1,Currentline);
								}
							} else {
								throw EDLUTFileException(13,63,3,1,Currentline);
							}
						} else {
							throw EDLUTFileException(13,64,3,1,Currentline);
						}
					} else {
						throw EDLUTFileException(13,65,3,1,Currentline);
					}
				} else {
					throw EDLUTFileException(13,66,3,1,Currentline);
				}
			} else {
				throw EDLUTFileException(13,67,3,1,Currentline);
			}
		} else {
			throw EDLUTFileException(13,68,3,1,Currentline);
		}
//----------------------------------------------------------------------	

  		//INTEGRATION METHOD
		this->integrationMethod_GPU=LoadIntegrationMethod_GPU::loadIntegrationMethod_GPU(fh, &Currentline, N_NeuronStateVariables, N_DifferentialNeuronState, N_TimeDependentNeuronState);

		//TIME DRIVEN STEP
		this->TimeDrivenStep_GPU = LoadTimeEvent_GPU::loadTimeEvent_GPU(fh, &Currentline);

	}
}

void LIFTimeDrivenModel_1_2_GPU::SynapsisEffect(int index, VectorNeuronState_GPU * state, Interconnection * InputConnection){

	switch (InputConnection->GetType()){
		case 0: {
			//gampa
			state->AuxStateCPU[0*state->GetSizeState() + index]+=1e-9f*InputConnection->GetWeight();
			break;
		}case 1:{
			//gnmda
			state->AuxStateCPU[1*state->GetSizeState() + index]+=1e-9f*InputConnection->GetWeight();
			break;
		}default :{
			printf("ERROR: LIFTimeDrivenModel_1_2 only support two kind of input synapses \n");
		}
	}
}

LIFTimeDrivenModel_1_2_GPU::LIFTimeDrivenModel_1_2_GPU(string NeuronTypeID, string NeuronModelID): TimeDrivenNeuronModel_GPU(NeuronTypeID, NeuronModelID), eexc(0), einh(0), erest(0), vthr(0), cm(0), texc(0), tinh(0),
		tref(0), grest(0){
}

LIFTimeDrivenModel_1_2_GPU::~LIFTimeDrivenModel_1_2_GPU(void){
	DeleteClassGPU2();
}

void LIFTimeDrivenModel_1_2_GPU::LoadNeuronModel() throw (EDLUTFileException){
	this->LoadNeuronModel(this->GetModelID()+".cfg");
}

VectorNeuronState * LIFTimeDrivenModel_1_2_GPU::InitializeState(){
	return this->GetVectorNeuronState();
}


InternalSpike * LIFTimeDrivenModel_1_2_GPU::ProcessInputSpike(PropagatedSpike *  InputSpike){
	Interconnection * inter = InputSpike->GetSource()->GetOutputConnectionAt(omp_get_thread_num(),InputSpike->GetTarget());

	Neuron * TargetCell = inter->GetTarget();

	int indexGPU =TargetCell->GetIndex_VectorNeuronState();

	VectorNeuronState_GPU * state = (VectorNeuronState_GPU *) this->InitialState;

	// Add the effect of the input spike
	this->SynapsisEffect(inter->GetTarget()->GetIndex_VectorNeuronState(), state, inter);

	return 0;
}

InternalSpike * LIFTimeDrivenModel_1_2_GPU::ProcessInputSpike(Interconnection * inter, Neuron * target, double time){
	int indexGPU =target->GetIndex_VectorNeuronState();

	VectorNeuronState_GPU * state = (VectorNeuronState_GPU *) this->InitialState;

	// Add the effect of the input spike
	this->SynapsisEffect(target->GetIndex_VectorNeuronState(), state, inter);

	return 0;
}


__global__ void LIFTimeDrivenModel_1_2_GPU_UpdateState(LIFTimeDrivenModel_1_2_GPU2 ** NeuronModel_GPU2, double CurrentTime){
	(*NeuronModel_GPU2)->UpdateState(CurrentTime);
}
		
bool LIFTimeDrivenModel_1_2_GPU::UpdateState(int index, VectorNeuronState * State, double CurrentTime){
	VectorNeuronState_GPU *state = (VectorNeuronState_GPU *) State;

	//----------------------------------------------
	if(prop.canMapHostMemory){
		LIFTimeDrivenModel_1_2_GPU_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
	}else{
		HANDLE_ERROR(hipMemcpy(state->AuxStateGPU,state->AuxStateCPU,4*state->SizeStates*sizeof(float),hipMemcpyHostToDevice));
		LIFTimeDrivenModel_1_2_GPU_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
		HANDLE_ERROR(hipMemcpy(state->InternalSpikeCPU,state->InternalSpikeGPU,state->SizeStates*sizeof(bool),hipMemcpyDeviceToHost));
	}


	if(this->GetVectorNeuronState()->Get_Is_Monitored()){
		HANDLE_ERROR(hipMemcpy(state->VectorNeuronStates,state->VectorNeuronStates_GPU,state->GetNumberOfVariables()*state->SizeStates*sizeof(float),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(state->LastUpdate,state->LastUpdateGPU,state->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(state->LastSpikeTime,state->LastSpikeTimeGPU,state->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
	}
 

	HANDLE_ERROR(hipEventRecord(stop, 0)); 
	HANDLE_ERROR(hipEventSynchronize(stop));


	memset(state->AuxStateCPU,0,N_TimeDependentNeuronState*state->SizeStates*sizeof(float));

	return false;

}

ostream & LIFTimeDrivenModel_1_2_GPU::PrintInfo(ostream & out){
	out << "- Leaky Time-Driven Model 1_2: " << this->GetModelID() << endl;

	out << "\tExc. Reversal Potential: " << this->eexc << "V\tInh. Reversal Potential: " << this->einh << "V\tResting potential: " << this->erest << "V" << endl;

	out << "\tFiring threshold: " << this->vthr << "V\tMembrane capacitance: " << this->cm << "nS\tExcitatory Time Constant: " << this->texc << "s" << endl;

	out << "\tInhibitory time constant: " << this->tinh << "s\tRefractory Period: " << this->tref << "s\tResting Conductance: " << this->grest << "nS" << endl;

	return out;
}	


void LIFTimeDrivenModel_1_2_GPU::InitializeStates(int N_neurons){

	VectorNeuronState_GPU * state = (VectorNeuronState_GPU *) this->InitialState;
	
	float initialization[] = {erest,0.0,0.0};
	state->InitializeStatesGPU(N_neurons, initialization, N_TimeDependentNeuronState);

	//INITIALIZE CLASS IN GPU
	this->InitializeClassGPU2(N_neurons);


	InitializeVectorNeuronState_GPU2();
}




__global__ void LIFTimeDrivenModel_1_2_GPU_InitializeClassGPU2(LIFTimeDrivenModel_1_2_GPU2 ** NeuronModel_GPU2, double new_elapsed_time,
		float eexc, float einh, float erest, float vthr, float cm, float texc, float tinh, float tref, float grest, 
		char const* integrationName, int N_neurons, void ** Buffer_GPU){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2)=new LIFTimeDrivenModel_1_2_GPU2(new_elapsed_time, eexc, einh, erest, 
			vthr, cm, texc, tinh, tref, grest, integrationName, N_neurons, Buffer_GPU);
	}
}

void LIFTimeDrivenModel_1_2_GPU::InitializeClassGPU2(int N_neurons){
	hipMalloc(&NeuronModel_GPU2, sizeof(LIFTimeDrivenModel_1_2_GPU2 **));
	
	char * integrationNameGPU;
	hipMalloc((void **)&integrationNameGPU,32*4);
	HANDLE_ERROR(hipMemcpy(integrationNameGPU,integrationMethod_GPU->GetType(),32*4,hipMemcpyHostToDevice));

	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceProperties( &prop, 0 ));	
	this->N_thread = 128;
	this->N_block=prop.multiProcessorCount*16;
	if((N_neurons+N_thread-1)/N_thread < N_block){
		N_block = (N_neurons+N_thread-1)/N_thread;
	}
	int Total_N_thread=N_thread*N_block;

	integrationMethod_GPU->InitializeMemoryGPU(N_neurons, Total_N_thread);

	LIFTimeDrivenModel_1_2_GPU_InitializeClassGPU2<<<1,1>>>(NeuronModel_GPU2,TimeDrivenStep_GPU, eexc, einh, erest, vthr, 
		cm, texc, tinh, tref, grest, integrationNameGPU, N_neurons, integrationMethod_GPU->Buffer_GPU);

	hipFree(integrationNameGPU);
}



__global__ void initializeVectorNeuronState_GPU2(LIFTimeDrivenModel_1_2_GPU2 ** NeuronModel_GPU2, float * AuxStateGPU, float * StateGPU, double * LastUpdateGPU, double * LastSpikeTimeGPU, bool * InternalSpikeGPU, int SizeStates){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2)->InitializeVectorNeuronState_GPU2(AuxStateGPU, StateGPU, LastUpdateGPU, LastSpikeTimeGPU, InternalSpikeGPU, SizeStates);
	}
}

void LIFTimeDrivenModel_1_2_GPU::InitializeVectorNeuronState_GPU2(){
	VectorNeuronState_GPU *state = (VectorNeuronState_GPU *) InitialState;
	initializeVectorNeuronState_GPU2<<<1,1>>>(NeuronModel_GPU2, state->AuxStateGPU, state->VectorNeuronStates_GPU, state->LastUpdateGPU, state->LastSpikeTimeGPU, state->InternalSpikeGPU, state->SizeStates);
}


__global__ void DeleteClass_GPU2(LIFTimeDrivenModel_1_2_GPU2 ** NeuronModel_GPU2){
	if(blockIdx.x==0 && threadIdx.x==0){
		delete (*NeuronModel_GPU2); 
	}
}


void LIFTimeDrivenModel_1_2_GPU::DeleteClassGPU2(){
    DeleteClass_GPU2<<<1,1>>>(NeuronModel_GPU2);
    hipFree(NeuronModel_GPU2);
}