#include "hip/hip_runtime.h"
/***************************************************************************
 *                           LIFTimeDrivenModel_1_4_GPU.cu                 *
 *                           -------------------                           *
 * copyright            : (C) 2011 by Francisco Naveros                    *
 * email                : fnaveros@atc.ugr.es                              *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

#include "../../include/neuron_model/LIFTimeDrivenModel_1_4_GPU.h"
#include "../../include/neuron_model/LIFTimeDrivenModel_1_4_GPU2.h"
#include "../../include/neuron_model/VectorNeuronState.h"
#include "../../include/neuron_model/VectorNeuronState_GPU.h"

#include <iostream>
#include <cmath>
#include <string>

#include "../../include/spike/EDLUTFileException.h"
#include "../../include/spike/Neuron.h"
#include "../../include/spike/InternalSpike.h"
#include "../../include/spike/PropagatedSpike.h"
#include "../../include/spike/Interconnection.h"

#include "../../include/simulation/Utils.h"

#include "../../include/openmp/openmp.h"

#include "../../include/hipError_t.h"
//Library for CUDA
#include <hip/hip_runtime_api.h>

void LIFTimeDrivenModel_1_4_GPU::LoadNeuronModel(string ConfigFile) throw (EDLUTFileException){
	FILE *fh;
	long Currentline = 0L;
	fh=fopen(ConfigFile.c_str(),"rt");
	if(fh){
		Currentline=1L;
		skip_comments(fh,Currentline);
		if(fscanf(fh,"%f",&this->eexc)==1){
			skip_comments(fh,Currentline);

			if (fscanf(fh,"%f",&this->einh)==1){
				skip_comments(fh,Currentline);

				if(fscanf(fh,"%f",&this->erest)==1){
					skip_comments(fh,Currentline);

					if(fscanf(fh,"%f",&this->vthr)==1){
						skip_comments(fh,Currentline);

						if(fscanf(fh,"%f",&this->cm)==1){
							skip_comments(fh,Currentline);

							if(fscanf(fh,"%f",&this->tampa)==1){
								skip_comments(fh,Currentline);

								if(fscanf(fh,"%f",&this->tnmda)==1){
									skip_comments(fh,Currentline);
									
									if(fscanf(fh,"%f",&this->tinh)==1){
										skip_comments(fh,Currentline);

										if(fscanf(fh,"%f",&this->tgj)==1){
											skip_comments(fh,Currentline);
											if(fscanf(fh,"%f",&this->tref)==1){
												skip_comments(fh,Currentline);

												if(fscanf(fh,"%f",&this->grest)==1){
													skip_comments(fh,Currentline);

													if(fscanf(fh,"%f",&this->fgj)==1){
														skip_comments(fh,Currentline);


														this->InitialState = (VectorNeuronState_GPU *) new VectorNeuronState_GPU(5);

													} else {
														throw EDLUTFileException(13,60,3,1,Currentline);
													}
												} else {
													throw EDLUTFileException(13,61,3,1,Currentline);
												}
											} else {
												throw EDLUTFileException(13,62,3,1,Currentline);
											}
										} else {
											throw EDLUTFileException(13,63,3,1,Currentline);
										}
									} else {
										throw EDLUTFileException(13,64,3,1,Currentline);
									}
								} else {
									throw EDLUTFileException(13,65,3,1,Currentline);
								}
							} else {
								throw EDLUTFileException(13,66,3,1,Currentline);
							}
						} else {
							throw EDLUTFileException(13,67,3,1,Currentline);
						}
					} else {
						throw EDLUTFileException(13,68,3,1,Currentline);
					}
				} else {
					throw EDLUTFileException(13,69,3,1,Currentline);
				}
			} else {
				throw EDLUTFileException(13,70,3,1,Currentline);
			}
		} else {
			throw EDLUTFileException(13,71,3,1,Currentline);
		}

  		//INTEGRATION METHOD
		this->integrationMethod_GPU=LoadIntegrationMethod_GPU::loadIntegrationMethod_GPU(fh, &Currentline, N_NeuronStateVariables, N_DifferentialNeuronState, N_TimeDependentNeuronState);

		//TIME DRIVEN STEP
		this->TimeDrivenStep_GPU = LoadTimeEvent_GPU::loadTimeEvent_GPU(fh, &Currentline);

	}
}

void LIFTimeDrivenModel_1_4_GPU::SynapsisEffect(int index, VectorNeuronState_GPU * state, Interconnection * InputConnection){

	switch (InputConnection->GetType()){
		case 0: {
			//gampa
			state->AuxStateCPU[0*state->GetSizeState() + index]+=InputConnection->GetWeight();
			break;
		}case 1:{
			//gnmda
			state->AuxStateCPU[1*state->GetSizeState() + index]+=InputConnection->GetWeight();
			break;
		}case 2:{
			//ginh
			state->AuxStateCPU[2*state->GetSizeState() + index]+=InputConnection->GetWeight();
			break;
		}case 3:{
			//ggj
			state->AuxStateCPU[3*state->GetSizeState() + index]+=InputConnection->GetWeight();
			break;
		}default :{
			printf("ERROR: LIFTimeDrivenModel_1_4 only support four kind of input synapses \n");
		}
	}
}

LIFTimeDrivenModel_1_4_GPU::LIFTimeDrivenModel_1_4_GPU(string NeuronTypeID, string NeuronModelID): TimeDrivenNeuronModel_GPU(NeuronTypeID, NeuronModelID), eexc(0), einh(0), erest(0), vthr(0), cm(0), tampa(0), tnmda(0), tinh(0), tgj(0),
		tref(0), grest(0){
}

LIFTimeDrivenModel_1_4_GPU::~LIFTimeDrivenModel_1_4_GPU(void){
	DeleteClassGPU2();
}

void LIFTimeDrivenModel_1_4_GPU::LoadNeuronModel() throw (EDLUTFileException){
	this->LoadNeuronModel(this->GetModelID()+".cfg");
}

VectorNeuronState * LIFTimeDrivenModel_1_4_GPU::InitializeState(){
	return this->GetVectorNeuronState();
}


InternalSpike * LIFTimeDrivenModel_1_4_GPU::ProcessInputSpike(PropagatedSpike *  InputSpike){
	Interconnection * inter = InputSpike->GetSource()->GetOutputConnectionAt(omp_get_thread_num(),InputSpike->GetTarget());

	Neuron * TargetCell = inter->GetTarget();

	int indexGPU =TargetCell->GetIndex_VectorNeuronState();

	VectorNeuronState_GPU * state = (VectorNeuronState_GPU *) this->InitialState;

	// Add the effect of the input spike
	this->SynapsisEffect(inter->GetTarget()->GetIndex_VectorNeuronState(), state, inter);

	return 0;
}


InternalSpike * LIFTimeDrivenModel_1_4_GPU::ProcessInputSpike(Interconnection * inter, Neuron * target, double time){
	int indexGPU =target->GetIndex_VectorNeuronState();

	VectorNeuronState_GPU * state = (VectorNeuronState_GPU *) this->InitialState;

	// Add the effect of the input spike
	this->SynapsisEffect(target->GetIndex_VectorNeuronState(), state, inter);

	return 0;
}


__global__ void LIFTimeDrivenModel_1_4_GPU_UpdateState(LIFTimeDrivenModel_1_4_GPU2 ** NeuronModel_GPU2, double CurrentTime){
	(*NeuronModel_GPU2)->UpdateState(CurrentTime);
}
		
bool LIFTimeDrivenModel_1_4_GPU::UpdateState(int index, VectorNeuronState * State, double CurrentTime){
	
	VectorNeuronState_GPU *state = (VectorNeuronState_GPU *) State;

	//----------------------------------------------
	if(prop.canMapHostMemory){
		LIFTimeDrivenModel_1_4_GPU_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
	}else{
		HANDLE_ERROR(hipMemcpy(state->AuxStateGPU,state->AuxStateCPU,4*state->SizeStates*sizeof(float),hipMemcpyHostToDevice));
		LIFTimeDrivenModel_1_4_GPU_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
		HANDLE_ERROR(hipMemcpy(state->InternalSpikeCPU,state->InternalSpikeGPU,state->SizeStates*sizeof(bool),hipMemcpyDeviceToHost));
	}


	if(this->GetVectorNeuronState()->Get_Is_Monitored()){
		HANDLE_ERROR(hipMemcpy(state->VectorNeuronStates,state->VectorNeuronStates_GPU,state->GetNumberOfVariables()*state->SizeStates*sizeof(float),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(state->LastUpdate,state->LastUpdateGPU,state->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(state->LastSpikeTime,state->LastSpikeTimeGPU,state->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
	}
 

	HANDLE_ERROR(hipEventRecord(stop, 0)); 
	HANDLE_ERROR(hipEventSynchronize(stop));


	memset(state->AuxStateCPU,0,N_TimeDependentNeuronState*state->SizeStates*sizeof(float));

	return false;

}

ostream & LIFTimeDrivenModel_1_4_GPU::PrintInfo(ostream & out){
	out << "- Leaky Time-Driven Model 1_4: " << this->GetModelID() << endl;

	out << "\tExc. Reversal Potential: " << this->eexc << "V\tInh. Reversal Potential: " << this->einh << "V\tResting potential: " << this->erest << "V" << endl;

	out << "\tFiring threshold: " << this->vthr << "V\tMembrane capacitance: " << this->cm << "nS\tAMPA Time Constant: " << this->tampa << "sNMDA Time Constant: " << this->tnmda << "s" << endl;

	out << "\tInhibitory time constant: " << this->tinh << "s\tGap junction time constant: " << this->tgj << "s\tRefractory Period: " << this->tref << "s\tResting Conductance: " << this->grest << "nS" << endl;

	return out;
}	


void LIFTimeDrivenModel_1_4_GPU::InitializeStates(int N_neurons){

	VectorNeuronState_GPU * state = (VectorNeuronState_GPU *) this->InitialState;
	
	float initialization[] = {erest,0.0,0.0,0.0,0.0};
	state->InitializeStatesGPU(N_neurons, initialization, N_TimeDependentNeuronState);

	//INITIALIZE CLASS IN GPU
	this->InitializeClassGPU2(N_neurons);


	InitializeVectorNeuronState_GPU2();
}




__global__ void LIFTimeDrivenModel_1_4_GPU_InitializeClassGPU2(LIFTimeDrivenModel_1_4_GPU2 ** NeuronModel_GPU2, double new_elapsed_time,
		float eexc,float einh,float erest,float vthr,float cm,float tampa,float tnmda,float tinh,float tgj,float tref,
		float grest,float fgj, char const* integrationName, int N_neurons, void ** Buffer_GPU){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2) = new LIFTimeDrivenModel_1_4_GPU2(new_elapsed_time, eexc,einh,erest,vthr,cm,
        tampa,tnmda,tinh,tgj,tref,grest,fgj,integrationName, N_neurons, Buffer_GPU);
	}
}
void LIFTimeDrivenModel_1_4_GPU::InitializeClassGPU2(int N_neurons){
	hipMalloc(&NeuronModel_GPU2, sizeof(LIFTimeDrivenModel_1_4_GPU2 **));
	
	char * integrationNameGPU;
	hipMalloc((void **)&integrationNameGPU,32*4);
	HANDLE_ERROR(hipMemcpy(integrationNameGPU,integrationMethod_GPU->GetType(),32*4,hipMemcpyHostToDevice));

	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceProperties( &prop, 0 ));	
	this->N_thread = 128;
	this->N_block=prop.multiProcessorCount*16;
	if((N_neurons+N_thread-1)/N_thread < N_block){
		N_block = (N_neurons+N_thread-1)/N_thread;
	}
	int Total_N_thread=N_thread*N_block;

	integrationMethod_GPU->InitializeMemoryGPU(N_neurons, Total_N_thread);

	LIFTimeDrivenModel_1_4_GPU_InitializeClassGPU2<<<1,1>>>(NeuronModel_GPU2,TimeDrivenStep_GPU, eexc,einh,erest,vthr,cm,tampa,
		tnmda,tinh,tgj,tref,grest,fgj,integrationNameGPU, N_neurons, integrationMethod_GPU->Buffer_GPU);

	hipFree(integrationNameGPU);
}



__global__ void initializeVectorNeuronState_GPU2(LIFTimeDrivenModel_1_4_GPU2 ** NeuronModel_GPU2, float * AuxStateGPU, float * StateGPU, double * LastUpdateGPU, double * LastSpikeTimeGPU, bool * InternalSpikeGPU, int SizeStates){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2)->InitializeVectorNeuronState_GPU2(AuxStateGPU, StateGPU, LastUpdateGPU, LastSpikeTimeGPU, InternalSpikeGPU, SizeStates);
	}
}

void LIFTimeDrivenModel_1_4_GPU::InitializeVectorNeuronState_GPU2(){
	VectorNeuronState_GPU *state = (VectorNeuronState_GPU *) InitialState;
	initializeVectorNeuronState_GPU2<<<1,1>>>(NeuronModel_GPU2, state->AuxStateGPU, state->VectorNeuronStates_GPU, state->LastUpdateGPU, state->LastSpikeTimeGPU, state->InternalSpikeGPU, state->SizeStates);
}


__global__ void DeleteClass_GPU2(LIFTimeDrivenModel_1_4_GPU2 ** NeuronModel_GPU2){
	if(blockIdx.x==0 && threadIdx.x==0){
		delete (*NeuronModel_GPU2); 
	}
}


void LIFTimeDrivenModel_1_4_GPU::DeleteClassGPU2(){
    DeleteClass_GPU2<<<1,1>>>(NeuronModel_GPU2);
    hipFree(NeuronModel_GPU2);
}




