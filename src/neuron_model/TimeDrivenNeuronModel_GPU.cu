/***************************************************************************
 *                           TimeDrivenNeuronModel_GPU.cpp                 *
 *                           -------------------                           *
 * copyright            : (C) 2012 by Francisco Naveros                    *
 * email                : fnaveros@atc.ugr.es                              *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

#include "../../include/neuron_model/TimeDrivenNeuronModel_GPU.h"
#include "../../include/neuron_model/TimeDrivenNeuronModel_GPU2.h"
#include "../../include/neuron_model/NeuronModel.h"
#include "../../include/neuron_model/VectorNeuronState.h"
#include "../../include/neuron_model/VectorNeuronState_GPU.h"


//Library for CUDA
#include "../../include/hipError_t.h"
#include <hip/hip_runtime_api.h>

#include <string>

TimeDrivenNeuronModel_GPU::TimeDrivenNeuronModel_GPU(string NeuronTypeID, string NeuronModelID): NeuronModel(NeuronTypeID, NeuronModelID), TimeDrivenStep_GPU(0.001) {
	// TODO Auto-generated constructor stub
}

TimeDrivenNeuronModel_GPU::~TimeDrivenNeuronModel_GPU() {
	delete integrationMethod_GPU;
	HANDLE_ERROR(hipEventDestroy(stop));
}

double TimeDrivenNeuronModel_GPU::GetTimeDrivenStep_GPU(){
	return TimeDrivenStep_GPU;
}


enum NeuronModelType TimeDrivenNeuronModel_GPU::GetModelType(){
	return TIME_DRIVEN_MODEL_GPU;
}

