#include "hip/hip_runtime.h"
/***************************************************************************
 *                           LIFTimeDrivenModel.cpp                        *
 *                           -------------------                           *
 * copyright            : (C) 2011 by Jesus Garrido                        *
 * email                : jgarrido@atc.ugr.es                              *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

//#include "../../include/neuron_model/LIFTimeDrivenModel_GPU.h"
//#include "../../include/neuron_model/NeuronState.h"
//
//#include <iostream>
//#include <cmath>
//#include <string>
//
//#include "../../include/spike/EDLUTFileException.h"
//#include "../../include/spike/Neuron.h"
//#include "../../include/spike/InternalSpike.h"
//#include "../../include/spike/PropagatedSpike.h"
//#include "../../include/spike/Interconnection.h"
//
//#include "../../include/simulation/Utils.h"

		#include "../../include/neuron_model/LIFTimeDrivenModel_CUDA.h"
		#include "../../include/hipError_t.h"
		//Library for CUDA
		#include <cutil_inline.h>



hipEvent_t synchronize;

void createSynchronize(){
	HANDLE_ERROR(hipEventCreate(&synchronize));
}

void synchronizeGPU_CPU(){
	HANDLE_ERROR(hipEventRecord(synchronize,0));
	HANDLE_ERROR(hipEventSynchronize(synchronize));
}

void destroySynchronize(){
	//HANDLE_ERROR(hipEventDestroy(synchronize));
}


__global__ void UpdateState(float * parameter, float * AuxStateGPU, float * StateGPU, double * LastUpdateGPU, double * LastSpikeTimeGPU, bool * InternalSpikeGPU, int SizeStates, double CurrentTime){
    float inv_param_4=1.e-9/parameter[4];

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int index4, index5;

    double elapsed_time =CurrentTime - LastUpdateGPU[index];
    float elapsed_time1 =elapsed_time;

	float exp_gampa = exp(-(elapsed_time1/parameter[5]));
	float exp_gnmda = exp(-(elapsed_time1/parameter[6]));
	float exp_ginh = exp(-(elapsed_time1/parameter[7]));
	float exp_ggj = exp(-(elapsed_time1/parameter[8]));

    while (index<SizeStates){
        index4 = index*4;
        index5 = index*5;

        LastSpikeTimeGPU[index]+=elapsed_time;
        double last_spike=LastSpikeTimeGPU[index];

        float vm = StateGPU[index5];
        float gampa = StateGPU[index5+1]+AuxStateGPU[index4];
        float gnmda = StateGPU[index5+2]+AuxStateGPU[index4+1];
        float ginh = StateGPU[index5+3]+AuxStateGPU[index4+2];
        float ggj = StateGPU[index5+4]+AuxStateGPU[index4+3];

        bool spike=false;

        if (last_spike > parameter[9]) {
            float iampa = gampa*(parameter[0]-vm);
            //float gnmdainf = 1.0/(1.0 + exp(-62.0*vm)*1.2/3.57);
			float gnmdainf = 1.0/(1.0 + exp(-62.0*vm)*0.336134453);
            float inmda = gnmda*gnmdainf*(parameter[0]-vm);
            float iinh = ginh*(parameter[1]-vm);
            vm = vm + elapsed_time * (iampa + inmda + iinh + parameter[10]* (parameter[2]-vm))*inv_param_4;

            float vm_cou = vm + parameter[11] * ggj;


            if (vm_cou > parameter[3]){
                LastSpikeTimeGPU[index]=0;
                spike = true;
                vm = parameter[2];
            }
        }

        InternalSpikeGPU[index]=spike;
	
        gampa *= exp_gampa;
        gnmda *= exp_gnmda;
        ginh *= exp_ginh;
        ggj *= exp_ggj;


        StateGPU[index5]=vm;
        StateGPU[index5+1]=gampa;
        StateGPU[index5+2]=gnmda;
        StateGPU[index5+3]=ginh;
        StateGPU[index5+4]=ggj;
        LastUpdateGPU[index]=CurrentTime;

        index+=blockDim.x*gridDim.x;
    }
}




void UpdateStateGPU(float * parameter, float * AuxStateGPU, float * AuxStateCPU, float * StateGPU, double * LastUpdateGPU, double * LastSpikeTimeGPU, bool * InternalSpikeGPU, bool * InternalSpikeCPU, int SizeStates, double CurrentTime){
	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceProperties( &prop, 0 ));

	int N_thread, N_block;
	
    //GPU can use MapHostMemory
    if(prop.canMapHostMemory){
        N_thread = 128;
		N_block=prop.multiProcessorCount*4;
		if((SizeStates+N_thread-1)/N_thread < N_block){
			N_block = (SizeStates+N_thread-1)/N_thread;
		}
		UpdateState<<<N_block,N_thread>>>(parameter, AuxStateGPU, StateGPU, LastUpdateGPU, LastSpikeTimeGPU, InternalSpikeGPU, SizeStates, CurrentTime);

    }

    //GPU can transfer memory and execute kernel at same time.
	else if(prop.deviceOverlap){
		N_thread = 128;
		N_block=prop.multiProcessorCount*4;
		if((SizeStates+N_thread-1)/N_thread < N_block){
			N_block = (SizeStates+N_thread-1)/N_thread;
		}

		const int N_Stream=4;
		
		hipStream_t stream[N_Stream];
		for (int i = 0; i < N_Stream; ++i){
			HANDLE_ERROR(hipStreamCreate(&stream[i]));
		}

		int size[N_Stream];
		int offset[N_Stream];

		int N_Stream_use;
		int aux=SizeStates/(N_thread*N_block);
		if(aux<N_Stream){
			if(aux==0){
				N_Stream_use=1;
			}else{
				N_Stream_use=aux;
			}
			for (int i = 0; i < N_Stream_use; ++i){
				offset[i]=i*N_thread*N_block;
				if(i==(N_Stream_use-1)){
					size[i]=SizeStates-offset[i];
				}else{
					size[i]=N_thread*N_block;
				}
			}
		}else{
			N_Stream_use=N_Stream;
			for (int i = 0; i < N_Stream_use; ++i){
				offset[i]=i*N_thread*N_block * (aux/N_Stream_use);
				if(i==(N_Stream_use-1)){
					size[i]=SizeStates-offset[i];
				}else{
					size[i]=N_thread*N_block * (aux/N_Stream_use);
				}
			}
		}

		HANDLE_ERROR(hipMemcpyAsync(AuxStateGPU, AuxStateCPU, sizeof(float)*4*size[0] , hipMemcpyHostToDevice, stream[0]));
		for (int i = 0; i < N_Stream_use; ++i) {
			if((i+1)<N_Stream_use){
				HANDLE_ERROR(hipMemcpyAsync(AuxStateGPU + offset[i+1] * 4, AuxStateCPU + offset[i+1] * 4, sizeof(float)*4*size[i+1] , hipMemcpyHostToDevice, stream[i+1]));
			}
			UpdateState<<<N_block,N_thread,0,stream[i]>>>(parameter, AuxStateGPU+ offset[i] * 4, StateGPU+ offset[i] * 5, LastUpdateGPU + offset[i], LastSpikeTimeGPU + offset[i], InternalSpikeGPU + offset[i], size[i], CurrentTime);
			HANDLE_ERROR(hipMemcpyAsync(InternalSpikeCPU + offset[i], InternalSpikeGPU + offset[i], sizeof(bool)*size[i],hipMemcpyDeviceToHost, stream[i]));
		}
		for (int i = 0; i < N_Stream; ++i){
			HANDLE_ERROR(hipStreamDestroy(stream[i]));
		}
	}
	
    //GPU uses memory transferences
	else{
		HANDLE_ERROR(hipMemcpy(AuxStateGPU,AuxStateCPU,4*SizeStates*sizeof(float),hipMemcpyHostToDevice));
		N_thread = 128;
		N_block=prop.multiProcessorCount*4;
		if((SizeStates+N_thread-1)/N_thread < N_block){
			N_block = (SizeStates+N_thread-1)/N_thread;
		}
		UpdateState<<<N_block,N_thread>>>(parameter, AuxStateGPU, StateGPU, LastUpdateGPU, LastSpikeTimeGPU, InternalSpikeGPU, SizeStates, CurrentTime);
		HANDLE_ERROR(hipMemcpy(InternalSpikeCPU,InternalSpikeGPU,SizeStates*sizeof(bool),hipMemcpyDeviceToHost));
	}

	HANDLE_ERROR(hipDeviceSynchronize());
}


void UpdateStateGPU(float * elapsed_time, float * parameter, float * AuxStateGPU, float * AuxStateCPU, float * StateGPU, double * LastUpdateGPU, double * LastSpikeTimeGPU, bool * InternalSpikeGPU, bool * InternalSpikeCPU, int SizeStates, double CurrentTime){
    hipEvent_t start, end;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&end));

    HANDLE_ERROR(hipEventRecord(start,0));	
	
	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceProperties( &prop, 0 ));

	int N_thread, N_block;

    //GPU can use MapHostMemory
    if(prop.canMapHostMemory){
        N_thread = 128;
        N_block=prop.multiProcessorCount*4;
        if((SizeStates+N_thread-1)/N_thread < N_block){
            N_block = (SizeStates+N_thread-1)/N_thread;
        }
        UpdateState<<<N_block,N_thread>>>(parameter, AuxStateGPU, StateGPU, LastUpdateGPU, LastSpikeTimeGPU, InternalSpikeGPU, SizeStates, CurrentTime);

    }

    //GPU can transfer memory and execute kernel at same time.
    else{ 
        if(prop.deviceOverlap){
		    N_thread = 128;
		    N_block=prop.multiProcessorCount*4;
		    if((SizeStates+N_thread-1)/N_thread < N_block){
			    N_block = (SizeStates+N_thread-1)/N_thread;
		    }

		    const int N_Stream=4;
    		
		    hipStream_t stream[N_Stream];
		    for (int i = 0; i < N_Stream; ++i){
			    HANDLE_ERROR(hipStreamCreate(&stream[i]));
		    }

		    int size[N_Stream];
		    int offset[N_Stream];

		    int N_Stream_use;
		    int aux=SizeStates/(N_thread*N_block);
		    if(aux<N_Stream){
			    if(aux==0){
				    N_Stream_use=1;
			    }else{
				    N_Stream_use=aux;
			    }
			    for (int i = 0; i < N_Stream_use; ++i){
				    offset[i]=i*N_thread*N_block;
				    if(i==(N_Stream_use-1)){
					    size[i]=SizeStates-offset[i];
				    }else{
					    size[i]=N_thread*N_block;
				    }
			    }
		    }else{
			    N_Stream_use=N_Stream;
			    for (int i = 0; i < N_Stream_use; ++i){
				    offset[i]=i*N_thread*N_block * (aux/N_Stream_use);
				    if(i==(N_Stream_use-1)){
					    size[i]=SizeStates-offset[i];
				    }else{
					    size[i]=N_thread*N_block * (aux/N_Stream_use);
				    }
			    }
		    }

		    HANDLE_ERROR(hipMemcpyAsync(AuxStateGPU, AuxStateCPU, sizeof(float)*4*size[0] , hipMemcpyHostToDevice, stream[0]));
		    for (int i = 0; i < N_Stream_use; ++i) {
			    if((i+1)<N_Stream_use){
				    HANDLE_ERROR(hipMemcpyAsync(AuxStateGPU + offset[i+1] * 4, AuxStateCPU + offset[i+1] * 4, sizeof(float)*4*size[i+1] , hipMemcpyHostToDevice, stream[i+1]));
			    }
			    UpdateState<<<N_block,N_thread,0,stream[i]>>>(parameter, AuxStateGPU+ offset[i] * 4, StateGPU+ offset[i] * 5, LastUpdateGPU + offset[i], LastSpikeTimeGPU + offset[i], InternalSpikeGPU + offset[i], size[i], CurrentTime);
			    HANDLE_ERROR(hipMemcpyAsync(InternalSpikeCPU + offset[i], InternalSpikeGPU + offset[i], sizeof(bool)*size[i],hipMemcpyDeviceToHost, stream[i]));
		    }
		    for (int i = 0; i < N_Stream; ++i){
			    HANDLE_ERROR(hipStreamDestroy(stream[i]));
		    }
	    }
    	
        //GPU uses memory transferences
	    else{
		    HANDLE_ERROR(hipMemcpy(AuxStateGPU,AuxStateCPU,4*SizeStates*sizeof(float),hipMemcpyHostToDevice));
		    N_thread = 128;
		    N_block=prop.multiProcessorCount*4;
		    if((SizeStates+N_thread-1)/N_thread < N_block){
			    N_block = (SizeStates+N_thread-1)/N_thread;
		    }
		    UpdateState<<<N_block,N_thread>>>(parameter, AuxStateGPU, StateGPU, LastUpdateGPU, LastSpikeTimeGPU, InternalSpikeGPU, SizeStates, CurrentTime);
		    HANDLE_ERROR(hipMemcpy(InternalSpikeCPU,InternalSpikeGPU,SizeStates*sizeof(bool),hipMemcpyDeviceToHost));
	    }
    }

	HANDLE_ERROR(hipDeviceSynchronize());

    HANDLE_ERROR(hipEventRecord(end,0));
    HANDLE_ERROR(hipEventSynchronize(end));
    HANDLE_ERROR(hipEventElapsedTime(elapsed_time,start,end));
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(end));
    //printf("Elapsed time: %f\n",*elapsed_time);
}



__global__ void UpdateStateRK(float * parameter, float * AuxStateGPU, float * StateGPU, double * LastUpdateGPU, double * LastSpikeTimeGPU, bool * InternalSpikeGPU, int SizeStates, double CurrentTime){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int index4, index5;

    double elapsed_time =CurrentTime - LastUpdateGPU[index];
	float elapsed_time1 =elapsed_time;

	float exp_gampa = exp(-(elapsed_time1/parameter[5]));
	float exp_gnmda = exp(-(elapsed_time1/parameter[6]));
	float exp_ginh = exp(-(elapsed_time1/parameter[7]));
	float exp_ggj = exp(-(elapsed_time1/parameter[8]));

	float exp_gampa2 = exp(-((elapsed_time1/2)/parameter[5]));
	float exp_gnmda2 = exp(-((elapsed_time1/2)/parameter[6]));
	float exp_ginh2 = exp(-((elapsed_time1/2)/parameter[7]));

    while (index<SizeStates){
        index4 = index*4;
        index5 = index*5;

        LastSpikeTimeGPU[index]+=elapsed_time;
        double last_spike=LastSpikeTimeGPU[index];

        float vm = StateGPU[index5];
        float gampa = StateGPU[index5+1]+AuxStateGPU[index4];
        float gnmda = StateGPU[index5+2]+AuxStateGPU[index4+1];
        float ginh = StateGPU[index5+3]+AuxStateGPU[index4+2];
        float ggj = StateGPU[index5+4]+AuxStateGPU[index4+3];

		float nextgampa = gampa * exp_gampa;
		float nextgnmda = gnmda * exp_gnmda;
		float nextginh = ginh * exp_ginh;
		float nextggj = ggj * exp_ggj;

        bool spike=false;

        if (last_spike > parameter[9]) {
			// 4th order Runge-Kutta terms
			// 1st term
			float iampa = gampa*(parameter[0]-vm);
			float gnmdainf = 1.0/(1.0 + exp(-62.0*vm)*1.2/3.57);
			float inmda = gnmda*gnmdainf*(parameter[0]-vm);
			float iinh = ginh*(parameter[1]-vm);
			
			float k1 = (iampa + inmda + iinh + parameter[10] * (parameter[2]-vm))*1.e-9/parameter[4];

			// 2nd term
			float gampaaux = gampa * exp_gampa2;
			float gnmdaaux = gnmda * exp_gnmda2;
			float ginhaux = ginh * exp_ginh2;
			float yaux = vm+(k1*elapsed_time/2);
			
			float iampaaux = gampaaux*(parameter[0]-yaux);
			float gnmdainfaux = 1.0/(1.0 + exp(-62.0*yaux)*1.2/3.57);
			float inmdaaux = gnmdaaux*gnmdainfaux*(parameter[0]-yaux);
			float iinhaux = ginhaux*(parameter[1]-yaux);
					
			float k2 = (iampaaux + inmdaaux + iinhaux + parameter[10] * (parameter[2] - yaux))*1.e-9/parameter[4];

			// 3rd term
			yaux = vm+(k2*elapsed_time/2);

			iampaaux = gampaaux*(parameter[0]-yaux);
			gnmdainfaux = 1.0/(1.0 + exp(-62.0*yaux)*1.2/3.57);
			inmdaaux = gnmdaaux*gnmdainfaux*(parameter[0]-yaux);
			iinhaux = ginhaux*(parameter[1]-yaux);
			
			float k3 = (iampaaux + inmdaaux + iinhaux + parameter[10] * (parameter[2] - yaux))*1.e-9/parameter[4];

			// 4rd term
			yaux = vm+(k3*elapsed_time);

			iampaaux = nextgampa*(parameter[0]-yaux);
			gnmdainfaux = 1.0/(1.0 + exp(-62.0*yaux)*1.2/3.57);
			inmdaaux = nextgampa*gnmdainfaux*(parameter[0]-yaux);
			iinhaux = nextginh*(parameter[1]-yaux);
			
			float k4 = (iampaaux + inmdaaux + iinhaux + parameter[10] * (parameter[2] - yaux))*1.e-9/parameter[4];

			vm = vm + (k1+2*k2+2*k3+k4)*elapsed_time/6;

			float vm_cou = vm + parameter[11] * ggj;


            if (vm_cou > parameter[3]){
                LastSpikeTimeGPU[index]=0;
                spike = true;
                vm = parameter[2];
            }
        }

        InternalSpikeGPU[index]=spike;
	
        gampa = nextgampa;
        gnmda = nextgnmda;
        ginh = nextginh;
        ggj = nextggj;


        StateGPU[index5]=vm;
        StateGPU[index5+1]=gampa;
        StateGPU[index5+2]=gnmda;
        StateGPU[index5+3]=ginh;
        StateGPU[index5+4]=ggj;
        LastUpdateGPU[index]=CurrentTime;

        index+=blockDim.x*gridDim.x;
    }
}



void UpdateStateRKGPU(float * parameter, float * AuxStateGPU, float * AuxStateCPU, float * StateGPU, double * LastUpdateGPU, double * LastSpikeTimeGPU, bool * InternalSpikeGPU, bool * InternalSpikeCPU, int SizeStates, double CurrentTime){
	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceProperties( &prop, 0 ));

	int N_thread, N_block;
	
    //GPU can use MapHostMemory
    if(prop.canMapHostMemory){
        N_thread = 128;
		N_block=prop.multiProcessorCount*4;
		if((SizeStates+N_thread-1)/N_thread < N_block){
			N_block = (SizeStates+N_thread-1)/N_thread;
		}
		UpdateStateRK<<<N_block,N_thread>>>(parameter, AuxStateGPU, StateGPU, LastUpdateGPU, LastSpikeTimeGPU, InternalSpikeGPU, SizeStates, CurrentTime);

    }

    //GPU can transfer memory and execute kernel at same time.
	else if(prop.deviceOverlap){
		N_thread = 128;
		N_block=prop.multiProcessorCount*4;
		if((SizeStates+N_thread-1)/N_thread < N_block){
			N_block = (SizeStates+N_thread-1)/N_thread;
		}

		const int N_Stream=4;
		
		hipStream_t stream[N_Stream];
		for (int i = 0; i < N_Stream; ++i){
			HANDLE_ERROR(hipStreamCreate(&stream[i]));
		}

		int size[N_Stream];
		int offset[N_Stream];

		int N_Stream_use;
		int aux=SizeStates/(N_thread*N_block);
		if(aux<N_Stream){
			if(aux==0){
				N_Stream_use=1;
			}else{
				N_Stream_use=aux;
			}
			for (int i = 0; i < N_Stream_use; ++i){
				offset[i]=i*N_thread*N_block;
				if(i==(N_Stream_use-1)){
					size[i]=SizeStates-offset[i];
				}else{
					size[i]=N_thread*N_block;
				}
			}
		}else{
			N_Stream_use=N_Stream;
			for (int i = 0; i < N_Stream_use; ++i){
				offset[i]=i*N_thread*N_block * (aux/N_Stream_use);
				if(i==(N_Stream_use-1)){
					size[i]=SizeStates-offset[i];
				}else{
					size[i]=N_thread*N_block * (aux/N_Stream_use);
				}
			}
		}

		HANDLE_ERROR(hipMemcpyAsync(AuxStateGPU, AuxStateCPU, sizeof(float)*4*size[0] , hipMemcpyHostToDevice, stream[0]));
		for (int i = 0; i < N_Stream_use; ++i) {
			if((i+1)<N_Stream_use){
				HANDLE_ERROR(hipMemcpyAsync(AuxStateGPU + offset[i+1] * 4, AuxStateCPU + offset[i+1] * 4, sizeof(float)*4*size[i+1] , hipMemcpyHostToDevice, stream[i+1]));
			}
			UpdateStateRK<<<N_block,N_thread,0,stream[i]>>>(parameter, AuxStateGPU+ offset[i] * 4, StateGPU+ offset[i] * 5, LastUpdateGPU + offset[i], LastSpikeTimeGPU + offset[i], InternalSpikeGPU + offset[i], size[i], CurrentTime);
			HANDLE_ERROR(hipMemcpyAsync(InternalSpikeCPU + offset[i], InternalSpikeGPU + offset[i], sizeof(bool)*size[i],hipMemcpyDeviceToHost, stream[i]));
		}
		for (int i = 0; i < N_Stream; ++i){
			HANDLE_ERROR(hipStreamDestroy(stream[i]));
		}
	}
	
    //GPU uses memory transferences
	else{
		HANDLE_ERROR(hipMemcpy(AuxStateGPU,AuxStateCPU,4*SizeStates*sizeof(float),hipMemcpyHostToDevice));
		N_thread = 128;
		N_block=prop.multiProcessorCount*4;
		if((SizeStates+N_thread-1)/N_thread < N_block){
			N_block = (SizeStates+N_thread-1)/N_thread;
		}
		UpdateStateRK<<<N_block,N_thread>>>(parameter, AuxStateGPU, StateGPU, LastUpdateGPU, LastSpikeTimeGPU, InternalSpikeGPU, SizeStates, CurrentTime);
		HANDLE_ERROR(hipMemcpy(InternalSpikeCPU,InternalSpikeGPU,SizeStates*sizeof(bool),hipMemcpyDeviceToHost));
	}

	HANDLE_ERROR(hipDeviceSynchronize());
}


void UpdateStateRKGPU(float * elapsed_time, float * parameter, float * AuxStateGPU, float * AuxStateCPU, float * StateGPU, double * LastUpdateGPU, double * LastSpikeTimeGPU, bool * InternalSpikeGPU, bool * InternalSpikeCPU, int SizeStates, double CurrentTime){
    hipEvent_t start, end;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&end));

    HANDLE_ERROR(hipEventRecord(start,0));	
	
	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceProperties( &prop, 0 ));

	int N_thread, N_block;

    //GPU can use MapHostMemory
    if(prop.canMapHostMemory){
        N_thread = 128;
        N_block=prop.multiProcessorCount*4;
        if((SizeStates+N_thread-1)/N_thread < N_block){
            N_block = (SizeStates+N_thread-1)/N_thread;
        }
        UpdateStateRK<<<N_block,N_thread>>>(parameter, AuxStateGPU, StateGPU, LastUpdateGPU, LastSpikeTimeGPU, InternalSpikeGPU, SizeStates, CurrentTime);

    }

    //GPU can transfer memory and execute kernel at same time.
    else{ 
        if(prop.deviceOverlap){
		    N_thread = 128;
		    N_block=prop.multiProcessorCount*4;
		    if((SizeStates+N_thread-1)/N_thread < N_block){
			    N_block = (SizeStates+N_thread-1)/N_thread;
		    }

		    const int N_Stream=4;
    		
		    hipStream_t stream[N_Stream];
		    for (int i = 0; i < N_Stream; ++i){
			    HANDLE_ERROR(hipStreamCreate(&stream[i]));
		    }

		    int size[N_Stream];
		    int offset[N_Stream];

		    int N_Stream_use;
		    int aux=SizeStates/(N_thread*N_block);
		    if(aux<N_Stream){
			    if(aux==0){
				    N_Stream_use=1;
			    }else{
				    N_Stream_use=aux;
			    }
			    for (int i = 0; i < N_Stream_use; ++i){
				    offset[i]=i*N_thread*N_block;
				    if(i==(N_Stream_use-1)){
					    size[i]=SizeStates-offset[i];
				    }else{
					    size[i]=N_thread*N_block;
				    }
			    }
		    }else{
			    N_Stream_use=N_Stream;
			    for (int i = 0; i < N_Stream_use; ++i){
				    offset[i]=i*N_thread*N_block * (aux/N_Stream_use);
				    if(i==(N_Stream_use-1)){
					    size[i]=SizeStates-offset[i];
				    }else{
					    size[i]=N_thread*N_block * (aux/N_Stream_use);
				    }
			    }
		    }

		    HANDLE_ERROR(hipMemcpyAsync(AuxStateGPU, AuxStateCPU, sizeof(float)*4*size[0] , hipMemcpyHostToDevice, stream[0]));
		    for (int i = 0; i < N_Stream_use; ++i) {
			    if((i+1)<N_Stream_use){
				    HANDLE_ERROR(hipMemcpyAsync(AuxStateGPU + offset[i+1] * 4, AuxStateCPU + offset[i+1] * 4, sizeof(float)*4*size[i+1] , hipMemcpyHostToDevice, stream[i+1]));
			    }
			    UpdateStateRK<<<N_block,N_thread,0,stream[i]>>>(parameter, AuxStateGPU+ offset[i] * 4, StateGPU+ offset[i] * 5, LastUpdateGPU + offset[i], LastSpikeTimeGPU + offset[i], InternalSpikeGPU + offset[i], size[i], CurrentTime);
			    HANDLE_ERROR(hipMemcpyAsync(InternalSpikeCPU + offset[i], InternalSpikeGPU + offset[i], sizeof(bool)*size[i],hipMemcpyDeviceToHost, stream[i]));
		    }
		    for (int i = 0; i < N_Stream; ++i){
			    HANDLE_ERROR(hipStreamDestroy(stream[i]));
		    }
	    }
    	
        //GPU uses memory transferences
	    else{
		    HANDLE_ERROR(hipMemcpy(AuxStateGPU,AuxStateCPU,4*SizeStates*sizeof(float),hipMemcpyHostToDevice));
		    N_thread = 128;
		    N_block=prop.multiProcessorCount*4;
		    if((SizeStates+N_thread-1)/N_thread < N_block){
			    N_block = (SizeStates+N_thread-1)/N_thread;
		    }
		    UpdateStateRK<<<N_block,N_thread>>>(parameter, AuxStateGPU, StateGPU, LastUpdateGPU, LastSpikeTimeGPU, InternalSpikeGPU, SizeStates, CurrentTime);
		    HANDLE_ERROR(hipMemcpy(InternalSpikeCPU,InternalSpikeGPU,SizeStates*sizeof(bool),hipMemcpyDeviceToHost));
	    }
    }

	HANDLE_ERROR(hipDeviceSynchronize());

    HANDLE_ERROR(hipEventRecord(end,0));
    HANDLE_ERROR(hipEventSynchronize(end));
    HANDLE_ERROR(hipEventElapsedTime(elapsed_time,start,end));
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(end));
    //printf("Elapsed time: %f\n",*elapsed_time);
}

void InformationGPU(){
	hipDeviceProp_t prop;
	int count;
	HANDLE_ERROR(hipGetDeviceCount( &count ));
	for (int i=0; i< count; i++) {
		hipGetDeviceProperties( &prop, i ) ;
		printf(" --- General Information for device %d ---\n", i );
		printf ( "Name: %s\n ", prop.name );
		printf( "Compute capability: %d.%d\n", prop.major, prop.minor );
		printf ( "Clock rate: %d\n", prop. clockRate );
		printf ( "Device copy overlap: ");
		if (prop.deviceOverlap)
			printf ( "Enabled\n" );
		else
			printf ( "Disabled\n" ) ;
		printf ( "Concurrent Kernels: ");
		if (prop.concurrentKernels)
			printf ( "Enabled\n" );
		else
			printf ( "Disabled\n" ) ;

		
		
		printf ( "Kernel execition timeout ");
		if (prop.kernelExecTimeoutEnabled)
			printf ( "Enabled\n" );
		else
			printf ( "Disabled\n" );
		
		printf("--- Memory Information for device %d ---\n", i );
		printf("Total global mem; %ld\n", prop. totalGlobalMem );
		printf("Total constant Mem: %ld\n", prop. totalConstMem );
		printf("Max mem pitch: %ld\n", prop.memPitch );
		printf("Texture Alignment: %ld\n", prop. textureAlignment );
		printf(" --- MP Information for device %d ---\n", i );
		printf ( "Multiprocessor count: %d\n",prop.multiProcessorCount );

		printf ( "Shared mem per mp: %ld\n", prop. sharedMemPerBlock );
		printf("Registers per rnp: %d\n", prop.regsPerBlock );
		printf("Threads in warp: %d\n", prop.warpSize ) ;
		printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);

		printf ( "Max thread dimensions: (%d, %d, %d) \n",
			prop. maxThreadsDim[0], prop.maxThreadsDim[1] ,
			prop.maxThreadsDim[2] );
		printf( "Max grid dimensions: (%d, %d, %d\n",
			prop.maxGridSize[0], prop.maxGridSize[1] ,
			prop.maxGridSize[2] );
		printf ( "\n" );
	}

}