#include "hip/hip_runtime.h"
/***************************************************************************
 *                           TimeDrivenPurkinjeCell_GPU.cu                 *
 *                           -------------------                           *
 * copyright            : (C) 2015 by Richard Carrill, Niceto Luque and    *
						  Francisco Naveros								   *
 * email                : rcarrillo@ugr.es, nluque@ugr.es and			   *
						  fnaveros@atc.ugr.es							   *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

#include "../../include/neuron_model/TimeDrivenPurkinjeCell_GPU.h"
#include "../../include/neuron_model/TimeDrivenPurkinjeCell_GPU2.h"
#include "../../include/neuron_model/VectorNeuronState.h"
#include "../../include/neuron_model/VectorNeuronState_GPU.h"

#include <iostream>
#include <cmath>
#include <string>

#include "../../include/spike/EDLUTFileException.h"
#include "../../include/spike/Neuron.h"
#include "../../include/spike/InternalSpike.h"
#include "../../include/spike/PropagatedSpike.h"
#include "../../include/spike/Interconnection.h"

#include "../../include/simulation/Utils.h"

#include "../../include/openmp/openmp.h"

#include "../../include/hipError_t.h"
//Library for CUDA
#include <hip/hip_runtime_api.h>

void TimeDrivenPurkinjeCell_GPU::LoadNeuronModel(string ConfigFile) throw (EDLUTFileException){
	FILE *fh;
	long Currentline = 0L;
	fh=fopen(ConfigFile.c_str(),"rt");
	if(fh){

		this->State = (VectorNeuronState_GPU *) new VectorNeuronState_GPU(N_NeuronStateVariables);

  		//INTEGRATION METHOD
		this->integrationMethod_GPU=LoadIntegrationMethod_GPU::loadIntegrationMethod_GPU(fh, &Currentline, N_NeuronStateVariables, N_DifferentialNeuronState, N_TimeDependentNeuronState);

		//TIME DRIVEN STEP
		this->TimeDrivenStep_GPU = LoadTimeEvent_GPU::loadTimeEvent_GPU(fh, &Currentline);

	}
}

void TimeDrivenPurkinjeCell_GPU::SynapsisEffect(int index, VectorNeuronState_GPU * state, Interconnection * InputConnection){
	state->AuxStateCPU[InputConnection->GetType()*state->GetSizeState() + index]+=1e-6f*InputConnection->GetWeight();
}

TimeDrivenPurkinjeCell_GPU::TimeDrivenPurkinjeCell_GPU(string NeuronTypeID, string NeuronModelID): TimeDrivenNeuronModel_GPU(NeuronTypeID, NeuronModelID), g_L(0.02f),
		g_Ca(0.001f), g_M(0.75f), Cylinder_length_of_the_soma(0.0015f), Radius_of_the_soma(0.0008f), Area(3.141592f*0.0015f*2.0f*0.0008f),
		inv_Area(1.0f/(3.141592f*0.0015f*2.0f*0.0008f)), Membrane_capacitance(1.0f), inv_Membrane_capacitance(1.0f/1.0f){

	eexc=0.0f;
	einh=-80.0f ;
	vthr= -35.0f;
	erest=-65.0f;
	texc=1.0f;
	inv_texc=1.0f/texc;
	tinh=2;
	inv_tinh=1.0f/tinh;
	tref=1.35f;
	tref_0_5=tref*0.5f;
	inv_tref_0_5=1.0f/tref_0_5;
	spkpeak=31.0f;
}

TimeDrivenPurkinjeCell_GPU::~TimeDrivenPurkinjeCell_GPU(void){
	DeleteClassGPU2();
}

void TimeDrivenPurkinjeCell_GPU::LoadNeuronModel() throw (EDLUTFileException){
	this->LoadNeuronModel(this->GetModelID()+".cfg");
}

VectorNeuronState * TimeDrivenPurkinjeCell_GPU::InitializeState(){
	return this->GetVectorNeuronState();
}


InternalSpike * TimeDrivenPurkinjeCell_GPU::ProcessInputSpike(Interconnection * inter, Neuron * target, double time){
	int indexGPU =target->GetIndex_VectorNeuronState();

	VectorNeuronState_GPU * state = (VectorNeuronState_GPU *) this->State;

	// Add the effect of the input spike
	this->SynapsisEffect(target->GetIndex_VectorNeuronState(), state, inter);

	return 0;
}


__global__ void TimeDrivenPurkinjeCell_GPU_UpdateState(TimeDrivenPurkinjeCell_GPU2 ** NeuronModel_GPU2, double CurrentTime){
	(*NeuronModel_GPU2)->UpdateState(CurrentTime);
}

		
bool TimeDrivenPurkinjeCell_GPU::UpdateState(int index, VectorNeuronState * State, double CurrentTime){
	VectorNeuronState_GPU *state = (VectorNeuronState_GPU *) State;

	//----------------------------------------------
	if(prop.canMapHostMemory){
		TimeDrivenPurkinjeCell_GPU_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
	}else{
		HANDLE_ERROR(hipMemcpy(state->AuxStateGPU,state->AuxStateCPU,4*state->SizeStates*sizeof(float),hipMemcpyHostToDevice));
		TimeDrivenPurkinjeCell_GPU_UpdateState<<<N_block,N_thread>>>(NeuronModel_GPU2, CurrentTime);
		HANDLE_ERROR(hipMemcpy(state->InternalSpikeCPU,state->InternalSpikeGPU,state->SizeStates*sizeof(bool),hipMemcpyDeviceToHost));
	}


	if(this->GetVectorNeuronState()->Get_Is_Monitored()){
		HANDLE_ERROR(hipMemcpy(state->VectorNeuronStates,state->VectorNeuronStates_GPU,state->GetNumberOfVariables()*state->SizeStates*sizeof(float),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(state->LastUpdate,state->LastUpdateGPU,state->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(state->LastSpikeTime,state->LastSpikeTimeGPU,state->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
	}
 

	HANDLE_ERROR(hipEventRecord(stop, 0)); 
	HANDLE_ERROR(hipEventSynchronize(stop));


	memset(state->AuxStateCPU,0,N_TimeDependentNeuronState*state->SizeStates*sizeof(float));

	return false;

}

ostream & TimeDrivenPurkinjeCell_GPU::PrintInfo(ostream & out){
	//out << "- Leaky Time-Driven Model 1_2: " << this->GetModelID() << endl;

	//out << "\tExc. Reversal Potential: " << this->eexc << "V\tInh. Reversal Potential: " << this->einh << "V\tResting potential: " << this->erest << "V" << endl;

	//out << "\tFiring threshold: " << this->vthr << "V\tMembrane capacitance: " << this->cm << "nS\tExcitatory Time Constant: " << this->texc << "s" << endl;

	//out << "\tInhibitory time constant: " << this->tinh << "s\tRefractory Period: " << this->tref << "s\tResting Conductance: " << this->grest << "nS" << endl;

	return out;
}	


void TimeDrivenPurkinjeCell_GPU::InitializeStates(int N_neurons, int OpenMPQueueIndex){

	//Select the correnpondent device. 
	HANDLE_ERROR(hipSetDevice(GPUsIndex[OpenMPQueueIndex % NumberOfGPUs]));  
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipGetDeviceProperties( &prop, GPUsIndex[OpenMPQueueIndex % NumberOfGPUs]));


	VectorNeuronState_GPU * state = (VectorNeuronState_GPU *) this->State;


	float alpha_ca=1.6f/(1+exp(-0.072f*(erest-5.0f)));

	float beta_ca=(0.02f*(erest+8.9f))/(exp((erest+8.9f)*0.2f)-1.0f);
	float inv_tau_ca=alpha_ca+beta_ca;

	float alpha_M=0.3f/(1+exp((-erest-2.0f)*0.2f));

	float beta_M=0.001f*exp((-erest-60.0f)*0.055555555555555f);
	float inv_tau_M=alpha_M+beta_M;

	//c_inf
	float c_inf=alpha_ca/inv_tau_ca;

	//M_inf
	float M_inf=alpha_M/inv_tau_M;

	float initialization[] = {erest,c_inf,M_inf,0.0f,0.0f};

	state->InitializeStatesGPU(N_neurons, initialization, N_TimeDependentNeuronState, prop);

	//INITIALIZE CLASS IN GPU
	this->InitializeClassGPU2(N_neurons);


	InitializeVectorNeuronState_GPU2();
}



__global__ void TimeDrivenPurkinjeCell_GPU_InitializeClassGPU2(TimeDrivenPurkinjeCell_GPU2 ** NeuronModel_GPU2, double new_elapsed_time,
	float new_g_L, float new_g_Ca, float new_g_M, float new_Cylinder_length_of_the_soma, float new_Radius_of_the_soma, float new_Area,
	float new_inv_Area, float new_Membrane_capacitance, float new_inv_Membrane_capacitance, float new_eexc,	float new_einh, 
	float new_vthr, float new_erest, float new_texc, float new_inv_texc, float new_tinh, float new_inv_tinh, float new_tref, 
	float new_tref_0_5, float new_inv_tref_0_5, float new_spkpeak, char const* integrationName, int N_neurons, void ** Buffer_GPU)
{
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2)=new TimeDrivenPurkinjeCell_GPU2(new_elapsed_time, new_g_L, new_g_Ca, new_g_M, new_Cylinder_length_of_the_soma, 
			new_Radius_of_the_soma, new_Area, new_inv_Area, new_Membrane_capacitance, new_inv_Membrane_capacitance, new_eexc, new_einh,	
			new_vthr, new_erest, new_texc, new_inv_texc, new_tinh, new_inv_tinh, new_tref, new_tref_0_5, new_inv_tref_0_5, new_spkpeak, 
			integrationName, N_neurons, Buffer_GPU);
	}
}

void TimeDrivenPurkinjeCell_GPU::InitializeClassGPU2(int N_neurons){
	hipMalloc(&NeuronModel_GPU2, sizeof(TimeDrivenPurkinjeCell_GPU2 **));
	
	char * integrationNameGPU;
	hipMalloc((void **)&integrationNameGPU,32*4);
	HANDLE_ERROR(hipMemcpy(integrationNameGPU,integrationMethod_GPU->GetType(),32*4,hipMemcpyHostToDevice));

	this->N_thread = 128;
	this->N_block=prop.multiProcessorCount*16;
	if((N_neurons+N_thread-1)/N_thread < N_block){
		N_block = (N_neurons+N_thread-1)/N_thread;
	}
	int Total_N_thread=N_thread*N_block;

	integrationMethod_GPU->InitializeMemoryGPU(N_neurons, Total_N_thread);


	TimeDrivenPurkinjeCell_GPU_InitializeClassGPU2<<<1,1>>>(NeuronModel_GPU2,TimeDrivenStep_GPU, g_L, g_Ca, g_M,Cylinder_length_of_the_soma, 
		Radius_of_the_soma, Area, inv_Area, Membrane_capacitance, inv_Membrane_capacitance, eexc, einh,	vthr, erest, texc, inv_texc, tinh, 
		inv_tinh, tref, tref_0_5, inv_tref_0_5, spkpeak, integrationNameGPU, N_neurons, integrationMethod_GPU->Buffer_GPU);

	hipFree(integrationNameGPU);
}



__global__ void initializeVectorNeuronState_GPU2(TimeDrivenPurkinjeCell_GPU2 ** NeuronModel_GPU2, float * AuxStateGPU, float * StateGPU, double * LastUpdateGPU, double * LastSpikeTimeGPU, bool * InternalSpikeGPU, int SizeStates){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*NeuronModel_GPU2)->InitializeVectorNeuronState_GPU2(AuxStateGPU, StateGPU, LastUpdateGPU, LastSpikeTimeGPU, InternalSpikeGPU, SizeStates);
	}
}

void TimeDrivenPurkinjeCell_GPU::InitializeVectorNeuronState_GPU2(){
	VectorNeuronState_GPU *state = (VectorNeuronState_GPU *) State;
	initializeVectorNeuronState_GPU2<<<1,1>>>(NeuronModel_GPU2, state->AuxStateGPU, state->VectorNeuronStates_GPU, state->LastUpdateGPU, state->LastSpikeTimeGPU, state->InternalSpikeGPU, state->SizeStates);
}


__global__ void DeleteClass_GPU2(TimeDrivenPurkinjeCell_GPU2 ** NeuronModel_GPU2){
	if(blockIdx.x==0 && threadIdx.x==0){
		delete (*NeuronModel_GPU2); 
	}
}


void TimeDrivenPurkinjeCell_GPU::DeleteClassGPU2(){
    DeleteClass_GPU2<<<1,1>>>(NeuronModel_GPU2);
    hipFree(NeuronModel_GPU2);
}


int TimeDrivenPurkinjeCell_GPU::CheckSynapseTypeNumber(int Type){
	if(Type<N_TimeDependentNeuronState && Type>=0){
		return Type;
	}else{
		cout<<"Neuron model "<<this->GetTypeID()<<", "<<this->GetModelID()<<" does not support input synapses of type "<<Type<<endl;
		return 0;
	}
}