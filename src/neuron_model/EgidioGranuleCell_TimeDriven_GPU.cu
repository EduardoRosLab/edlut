#include "hip/hip_runtime.h"
/***************************************************************************
 *                           EgidioGranuleCell_TimeDriven_GPU.cu           *
 *                           -------------------                           *
 * copyright            : (C) 2013 by Francisco Naveros                    *
 * email                : fnaveros@atc.ugr.es                              *
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 3 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

#include "../../include/neuron_model/EgidioGranuleCell_TimeDriven_GPU.h"
#include "../../include/neuron_model/EgidioGranuleCell_TimeDriven_GPU2.h"
#include "../../include/neuron_model/VectorNeuronState.h"
#include "../../include/neuron_model/VectorNeuronState_GPU.h"

#include <iostream>
#include <cmath>
#include <string>


#include "../../include/spike/EDLUTFileException.h"
#include "../../include/spike/Neuron.h"
#include "../../include/spike/InternalSpike.h"
#include "../../include/spike/PropagatedSpike.h"
#include "../../include/spike/Interconnection.h"

#include "../../include/simulation/Utils.h"

#include "../../include/hipError_t.h"
//Library for CUDA
#include <hip/hip_runtime_api.h>

void EgidioGranuleCell_TimeDriven_GPU::LoadNeuronModel(string ConfigFile) throw (EDLUTFileException){
	FILE *fh;
	long Currentline = 0L;
	fh=fopen(ConfigFile.c_str(),"rt");
	if(fh){

		Currentline=1L;
		skip_comments(fh,Currentline);
		if(fscanf(fh,"%f",&this->gMAXNa_f)==1){
			skip_comments(fh,Currentline);

			if (fscanf(fh,"%f",&this->gMAXNa_r)==1){
				skip_comments(fh,Currentline);

				if(fscanf(fh,"%f",&this->gMAXNa_p)==1){
					skip_comments(fh,Currentline);

					if(fscanf(fh,"%f",&this->gMAXK_V)==1){
						skip_comments(fh,Currentline);

						if(fscanf(fh,"%f",&this->gMAXK_A)==1){
							skip_comments(fh,Currentline);

							if(fscanf(fh,"%f",&this->gMAXK_IR)==1){
								skip_comments(fh,Currentline);

								if(fscanf(fh,"%f",&this->gMAXK_Ca)==1){
									skip_comments(fh,Currentline);

									if(fscanf(fh,"%f",&this->gMAXCa)==1){
										skip_comments(fh,Currentline);

										if(fscanf(fh,"%f",&this->gMAXK_sl)==1){
											skip_comments(fh,Currentline);

											this->InitialState = (VectorNeuronState_GPU *) new VectorNeuronState_GPU(17);

										}
//NEW CODE------------------------------------------------------------------------------									
										else {
											throw EDLUTFileException(13,60,3,1,Currentline);
										}
									} else {
										throw EDLUTFileException(13,61,3,1,Currentline);
									}
								} else {
									throw EDLUTFileException(13,62,3,1,Currentline);
								}
							} else {
								throw EDLUTFileException(13,63,3,1,Currentline);
							}
						} else {
							throw EDLUTFileException(13,64,3,1,Currentline);
						}
					} else {
						throw EDLUTFileException(13,65,3,1,Currentline);
					}
				} else {
					throw EDLUTFileException(13,66,3,1,Currentline);
				}
			} else {
				throw EDLUTFileException(13,67,3,1,Currentline);
			}
		} else {
			throw EDLUTFileException(13,68,3,1,Currentline);
		}
//-------------------------------------------------------------------------------------

		//INTEGRATION METHOD
		this->integrationMethod_GPU=LoadIntegrationMethod_GPU::loadIntegrationMethod_GPU(fh, &Currentline, N_NeuronStateVariables, N_DifferentialNeuronState, N_TimeDependentNeuronState);

		//TIME DRIVEN STEP
		this->TimeDrivenStep_GPU = LoadTimeEvent_GPU::loadTimeEvent_GPU(fh, &Currentline);
	}
}

void EgidioGranuleCell_TimeDriven_GPU::SynapsisEffect(int index, VectorNeuronState_GPU * state, Interconnection * InputConnection){

	switch (InputConnection->GetType()){
		case 0: {
			state->AuxStateCPU[0*state->GetSizeState() + index]+=1e-9f*InputConnection->GetWeight();
			break;
		}case 1:{
			state->AuxStateCPU[1*state->GetSizeState() + index]+=1e-9f*InputConnection->GetWeight();
			break;
		}default :{
			printf("ERROR: EgidioGranuleCell_TimeDriven only support two kind of input synapses \n");
		}
	}
}



EgidioGranuleCell_TimeDriven_GPU::EgidioGranuleCell_TimeDriven_GPU(string NeuronTypeID, string NeuronModelID): TimeDrivenNeuronModel_GPU(NeuronTypeID, NeuronModelID), gMAXNa_f(0.0f), gMAXNa_r(0.0f), gMAXNa_p(0.0f), gMAXK_V(0.0f), gMAXK_A(0.0f), gMAXK_IR(0.0f), gMAXK_Ca(0.0f),
		gMAXCa(0.0f), gMAXK_sl(0.0f), gLkg1(5.68e-5f), gLkg2(2.17e-5f), VNa(87.39f), VK(-84.69f), VLkg1(-58.0f), VLkg2(-65.0f), V0_xK_Ai(-46.7f),
		K_xK_Ai(-19.8f), V0_yK_Ai(-78.8f), K_yK_Ai(8.4f), V0_xK_sli(-30.0f), B_xK_sli(6.0f), F(96485.309f), A(1e-04f), d(0.2f), betaCa(1.5f),
		Ca0(1e-04f), R(8.3134f), cao(2.0f), Cm(1.0e-3f), temper(30.0f), Q10_20 ( pow(3,((temper-20.0f)/10.0f))), Q10_22 ( pow(3,((temper-22.0f)/10.0f))),
		Q10_30 ( pow(3,((temper-30.0f)/10.0f))), Q10_6_3 ( pow(3,((temper-6.3f)/10.0f))),	/*I_inj_abs(11e-12f)*/I_inj_abs(0.0f),
		I_inj(-I_inj_abs*1000.0f/299.26058e-8f), eexc(0.0f), einh(-80.0f), texc(0.5f), tinh(10.0f), vthr(-0.25f){
}

EgidioGranuleCell_TimeDriven_GPU::~EgidioGranuleCell_TimeDriven_GPU(void){
	DeleteClassGPU();
}

void EgidioGranuleCell_TimeDriven_GPU::LoadNeuronModel() throw (EDLUTFileException){
	this->LoadNeuronModel(this->GetModelID()+".cfg");
}


VectorNeuronState * EgidioGranuleCell_TimeDriven_GPU::InitializeState(){
	return this->GetVectorNeuronState();
}


InternalSpike * EgidioGranuleCell_TimeDriven_GPU::ProcessInputSpike(PropagatedSpike *  InputSpike){
	Interconnection * inter = InputSpike->GetSource()->GetOutputConnectionAt(InputSpike->GetTarget());

	Neuron * TargetCell = inter->GetTarget();

	int indexGPU =TargetCell->GetIndex_VectorNeuronState();

	VectorNeuronState_GPU * state = (VectorNeuronState_GPU *) this->InitialState;

	// Add the effect of the input spike
	this->SynapsisEffect(inter->GetTarget()->GetIndex_VectorNeuronState(), state, inter);

	return 0;
}

InternalSpike * EgidioGranuleCell_TimeDriven_GPU::ProcessInputSpike(Interconnection * inter, Neuron * target, double time){
	int indexGPU =target->GetIndex_VectorNeuronState();

	VectorNeuronState_GPU * state = (VectorNeuronState_GPU *) this->InitialState;

	// Add the effect of the input spike
	this->SynapsisEffect(target->GetIndex_VectorNeuronState(), state, inter);

	return 0;
}



__global__ void EgidioGranuleCell_TimeDriven_GPU_UpdateState(int size, int offset, TimeDrivenNeuronModel_GPU2 ** timeDrivenNeuronModel_GPU2, float * AuxStateGPU, float * StateGPU, double * LastUpdateGPU, double * LastSpikeTimeGPU, bool * InternalSpikeGPU, int SizeStates, double CurrentTime){
	int index = offset + blockIdx.x * blockDim.x + threadIdx.x;
	while (index < (offset + size) && index<SizeStates){
		(*timeDrivenNeuronModel_GPU2)->UpdateState(index, AuxStateGPU, StateGPU, LastUpdateGPU, LastSpikeTimeGPU, InternalSpikeGPU, SizeStates, CurrentTime);
		index+=blockDim.x*gridDim.x;
	}
}
		
bool EgidioGranuleCell_TimeDriven_GPU::UpdateState(int index, VectorNeuronState * State, double CurrentTime){
	
	VectorNeuronState_GPU *state = (VectorNeuronState_GPU *) State;

	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceProperties( &prop, 0 ));
	if(prop.canMapHostMemory && true){
		int size=N_block*N_thread;
		int offset=0;
		while (offset<state->SizeStates){
			EgidioGranuleCell_TimeDriven_GPU_UpdateState<<<N_block,N_thread>>>(size,offset,timeDrivenNeuronModel_GPU2, state->AuxStateGPU, state->VectorNeuronStates_GPU, state->LastUpdateGPU, state->LastSpikeTimeGPU, state->InternalSpikeGPU, state->SizeStates, CurrentTime);
			offset+=size;
		}
	}else{
		HANDLE_ERROR(hipMemcpy(state->AuxStateGPU,state->AuxStateCPU,4*state->SizeStates*sizeof(float),hipMemcpyHostToDevice));
		int size=N_block*N_thread*2;
		int offset=0;
		while (offset<state->SizeStates){
			EgidioGranuleCell_TimeDriven_GPU_UpdateState<<<N_block,N_thread>>>(size, offset, timeDrivenNeuronModel_GPU2, state->AuxStateGPU, state->VectorNeuronStates_GPU, state->LastUpdateGPU, state->LastSpikeTimeGPU, state->InternalSpikeGPU, state->SizeStates, CurrentTime);
			offset+=size;
		}
		HANDLE_ERROR(hipMemcpy(state->InternalSpikeCPU,state->InternalSpikeGPU,state->SizeStates*sizeof(bool),hipMemcpyDeviceToHost));
	}


	if(this->GetVectorNeuronState()->Get_Is_Monitored()){
		HANDLE_ERROR(hipMemcpy(state->VectorNeuronStates,state->VectorNeuronStates_GPU,state->GetNumberOfVariables()*state->SizeStates*sizeof(float),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(state->LastUpdate,state->LastUpdateGPU,state->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(state->LastSpikeTime,state->LastSpikeTimeGPU,state->SizeStates*sizeof(double),hipMemcpyDeviceToHost));
	}

	HANDLE_ERROR(hipEventRecord(stop, 0)); 
	HANDLE_ERROR(hipEventSynchronize(stop));

	memset(state->AuxStateCPU,0,N_TimeDependentNeuronState*state->SizeStates*sizeof(float));

	return false;

}


ostream & EgidioGranuleCell_TimeDriven_GPU::PrintInfo(ostream & out){
	return out;
}	


void EgidioGranuleCell_TimeDriven_GPU::InitializeStates(int N_neurons){

	VectorNeuronState_GPU * state = (VectorNeuronState_GPU *) this->InitialState;

	//Initial State
	float xNa_f=0.00047309535f;
	float yNa_f=1.0f;
	float xNa_r=0.00013423511f;
	float yNa_r=0.96227829f;
	float xNa_p=0.00050020111f;
	float xK_V=0.010183001f;
	float xK_A=0.15685486f;
	float yK_A=0.53565367f;
	float xK_IR=0.37337035f;
	float xK_Ca=0.00012384122f;
	float xCa=0.0021951104f;
	float yCa=0.89509747f;
	float xK_sl=0.00024031171f;
	float Ca=Ca0;
	float V=-80.0f;
	float gexc=0.0f;
	float ginh=0.0f;

	//Initialize neural state variables.
	float initialization[] = {xNa_f,yNa_f,xNa_r,yNa_r,xNa_p,xK_V,xK_A,yK_A,xK_IR,xK_Ca,xCa,yCa,xK_sl,Ca,V,gexc,ginh};
	state->InitializeStatesGPU(N_neurons, initialization, N_TimeDependentNeuronState);

	//INITIALIZE CLASS IN GPU
	this->InitializeClassGPU(N_neurons);
}


__global__ void EgidioGranuleCell_TimeDriven_GPU2_InitializeClassGPU(TimeDrivenNeuronModel_GPU2 ** timeDrivenNeuronModel_GPU2,
		float gMAXNa_f, float gMAXNa_r, float gMAXNa_p, float gMAXK_V,
		float gMAXK_A,float gMAXK_IR,float gMAXK_Ca,float gMAXCa,float gMAXK_sl, char const* integrationName, int N_neurons, int Total_N_thread, void ** Buffer_GPU){
	if(blockIdx.x==0 && threadIdx.x==0){
		(*timeDrivenNeuronModel_GPU2) = (EgidioGranuleCell_TimeDriven_GPU2 *) new EgidioGranuleCell_TimeDriven_GPU2(gMAXNa_f, gMAXNa_r, gMAXNa_p, gMAXK_V,
			gMAXK_A,gMAXK_IR,gMAXK_Ca,gMAXCa,gMAXK_sl,integrationName, N_neurons, Total_N_thread, Buffer_GPU);
	}
}


void EgidioGranuleCell_TimeDriven_GPU::InitializeClassGPU(int N_neurons){
	hipMalloc(&timeDrivenNeuronModel_GPU2, sizeof(TimeDrivenNeuronModel_GPU2 **));
	
	char * integrationNameGPU;
	hipMalloc((void **)&integrationNameGPU,32*4);
	HANDLE_ERROR(hipMemcpy(integrationNameGPU,integrationMethod_GPU->GetType(),32*4,hipMemcpyHostToDevice));

	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceProperties( &prop, 0 ));	
	this->N_thread = 128;
	this->N_block=prop.multiProcessorCount*4;
	if((N_neurons+N_thread-1)/N_thread < N_block){
		N_block = (N_neurons+N_thread-1)/N_thread;
	}
	int Total_N_thread=N_thread*N_block;

	integrationMethod_GPU->InitializeMemoryGPU(N_neurons, Total_N_thread);

	
	EgidioGranuleCell_TimeDriven_GPU2_InitializeClassGPU<<<1,1>>>(timeDrivenNeuronModel_GPU2,gMAXNa_f, gMAXNa_r, gMAXNa_p, gMAXK_V,
			gMAXK_A,gMAXK_IR,gMAXK_Ca,gMAXCa,gMAXK_sl,integrationNameGPU, N_neurons,Total_N_thread, integrationMethod_GPU->Buffer_GPU);

	hipFree(integrationNameGPU);
}



__global__ void EgidioGranuleCell_TimeDriven_GPU_DeleteClassGPU(TimeDrivenNeuronModel_GPU2 ** timeDrivenNeuronModel_GPU2){
	if(blockIdx.x==0 && threadIdx.x==0){
		delete (*timeDrivenNeuronModel_GPU2); 
	}
}

void EgidioGranuleCell_TimeDriven_GPU::DeleteClassGPU(){
    EgidioGranuleCell_TimeDriven_GPU_DeleteClassGPU<<<1,1>>>(timeDrivenNeuronModel_GPU2);
    hipFree(timeDrivenNeuronModel_GPU2);
}